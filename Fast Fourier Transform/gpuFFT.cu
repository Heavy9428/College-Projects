#include "hip/hip_runtime.h"
//***************************************
//FFT Parallel
//Name: KEVIN ELLIS, CARLOS PLACENCIA, AND MATT TREBING
//Midwestern State University
//GPU Spring 2018
//(04/23/2018)
//****************************************
/* 
	This program computes the FFT in parallel using cuda
	Runs the program X amount of times and computes the time
	for each iteration. Then averages the time taken
	Each process with get a subtable from the table. Then calculate 
	even and odd for each coeffiencet X then P0 will gather these values
	and print them to the file
	
    TURING COMPILE: ssh gpu
					/opt/cuda-8.0/bin/nvcc gpuFFT.cu -o a
    TURING RUN: ./a

	TACC COMPILE:   module load cuda
                    nvcc -arch=compute_35 -code=sm_35 gpuFFT.cu -o a.out
	TACC RUN: sbatch gpuScript
*/
#include <stdio.h>
#include <hip/hip_complex.h>  //Cuda Complex numbers!!!! Very useful here!
#include <math.h>	//for cos() and sin()
#include "timer.h" //to use timer

#define PI 3.14159265
#define bigN 16384 //Problem Size
#define numBlocks 4 //How many Block do I wanna use??     -> Adaptive is bigN / 1024
#define numThreadsPerBlockx 1024 //How many threads per block ?? -> If numBlocks * numThreads > bigN
								//Then you need to lower this number to bigN / numBlocks

#define howmanytimesavg 3 //How many times do I wanna run for the AVG?
#define howmanytoprint 8 //How many Xi's do I wanna print?

__global__
void KernalFFT(hipDoubleComplex * inNumbers, hipDoubleComplex * outResults)
{
	//get the threads ID
	int threadID = threadIdx.x + blockDim.x * blockIdx.x;
	
	//Create Accumulators for the sum of the EVEN and ODD parts
	//I.E starting as (0 + 0i)
	hipDoubleComplex sumOfEven = make_hipDoubleComplex(0, 0); 
	hipDoubleComplex sumOfOdd = make_hipDoubleComplex(0, 0);
	
	//LOOP that goes 0 to N/2 - 1: [look at FFT formula]
	for (int i = 0; i <= (bigN / 2) - 1; i++) {
		//==========================EVEN PART STARTS HERE================================
		hipDoubleComplex evenFromTable = inNumbers[2 * i];   // 2n gives all the even numbers

		double factorEven = (2*PI * (2*i) * threadID) / bigN; //Calculates the even factor for Cos() and Sin()
															 //*********Reduces computational time*********

		double realPartEven = cos(factorEven); //COS part of the equation for the REAL PART
		double imagPartEven = -1 * sin(factorEven); //SIN part of the equation for the IMAG PART

		hipDoubleComplex wholePartEven = make_hipDoubleComplex(realPartEven, imagPartEven);// CREATES: realPartEven + imagPartEven * I
		hipDoubleComplex resultEven = hipCmul(evenFromTable, wholePartEven);				//MULTIPLIES the actual numberfrom table with the number created from COS and SIN
																					   //EX)     (2.6 + 1i) * (1 - 0)    ----> resultEven	
		sumOfEven = hipCadd(resultEven, sumOfEven); //finally... accumulate all the EVEN numbers up for later
		//==============================================================================

		//==========================ODD PART STARTS HERE================================
		hipDoubleComplex oddromTable = inNumbers[2 * i + 1];   // 2n + 1 gives all the odd numbers

		double factorOdd = (2*PI * (2*i + 1) * threadID) / bigN; //Calculates the odd factor for Cos() and Sin()
															    //*********Reduces computational time*********

		double realPartOdd = cos(factorOdd); //COS part of the equation for the REAL PART
		double imagPartOdd = -1 * sin(factorOdd); //SIN part of the equation for the IMAG PART

		hipDoubleComplex wholePartOdd = make_hipDoubleComplex(realPartOdd, imagPartOdd);// CREATES: realPartEven + imagPartEven * I
		hipDoubleComplex resultOdd = hipCmul(oddromTable, wholePartOdd);				//MULTIPLIES the actual numberfrom table with the number created from COS and SIN
																					   //EX)     (2.6 + 1i) * (1 - 0)    ----> resultOdd	

		sumOfOdd = hipCadd(resultOdd, sumOfOdd); //finally... accumulate all the EVEN numbers up for later
		//==============================================================================

	}
	outResults[threadID] = hipCadd(sumOfEven, sumOfOdd);
} 


int main()
{	
	double avgtime = 0;
	int h;
	FILE *outfile;
	outfile = fopen("ParallelVersionOutput.txt", "w"); //oepn from current directory

	for(h = 0;h < howmanytimesavg; h++ )
	{
		double start,finish; //For time
											
		hipDoubleComplex * tableValues = (hipDoubleComplex*) malloc(bigN * sizeof(hipDoubleComplex)); //allocate memeory to store table of signals
    	hipDoubleComplex * returnedFFT = (hipDoubleComplex*) malloc(bigN * sizeof(hipDoubleComplex)); //allocate memeory to store what kernal does
    	hipDoubleComplex * tableValuesd;  //device pointer
    	hipDoubleComplex * returnedFFTd; //device pointer

		//LOAD FIRST 8 VALUES OF TABLE UP
		tableValues[0] = make_hipDoubleComplex(3.6, 2.6); // 3.6 + 2.6i
		tableValues[1] = make_hipDoubleComplex(2.9, 6.3); // 2.9 + 6.3i
		tableValues[2] = make_hipDoubleComplex(5.6, 4.0); // ...
		tableValues[3] = make_hipDoubleComplex(4.8, 9.1); // ...
		tableValues[4] = make_hipDoubleComplex(3.3, 0.4); // ... 
		tableValues[5] = make_hipDoubleComplex(5.9, 4.8); // ...
		tableValues[6] = make_hipDoubleComplex(5.0, 2.6); // ...
		tableValues[7] = make_hipDoubleComplex(4.3, 4.1); // 4.3 + 4.1i

		//EVERYTHING AFTER ROW 8 IS 0
		if(bigN > 8)
		{
			for(int i = 8; i < bigN; i++)
			{
				tableValues[i] = make_hipDoubleComplex(0, 0); 
			}
		}

		//allocate memory on the GPU for table of values & what the FFT will return'
		double memorySize = bigN * sizeof(hipDoubleComplex);
		hipMalloc((void **)&tableValuesd, memorySize);
		hipMalloc((void **)&returnedFFTd, memorySize);

		//copy the table from Host to GPU, so that the GPU can perfom FFT
		hipMemcpy(tableValuesd, tableValues, memorySize, hipMemcpyHostToDevice);

		GET_TIME(start); //start the timer
		// HOW MANY TIMES DO I NEED TO CALL KERNAL????????????
		for(int r = 0; r < bigN / (numBlocks * numThreadsPerBlockx); r++){
			dim3 dimGrid( numBlocks ,1); //Set up Grid for this run
			dim3 dimBlock(numThreadsPerBlockx,1); //Set up Blocks for this run
			KernalFFT<<<dimGrid, dimBlock>>>(tableValuesd, returnedFFTd); //call kernal....passing it the table & where to store FFT results
			hipDeviceSynchronize();
		}
		GET_TIME(finish); //stop timer

		//GET THE RESULT FROM THE KERNAL
		hipMemcpy(returnedFFT, returnedFFTd, memorySize, hipMemcpyDeviceToHost);

		//FREE MEMORY ON THE GPU...
		hipFree(tableValuesd); 
		hipFree(returnedFFTd);

		//Print the first 8 K's
		fprintf(outfile," \n\nTOTAL PROCESSED SAMPLES : %d\n",bigN);
		for(int i = 0; i < howmanytoprint; i++){
			fprintf(outfile,"================================\n");
			fprintf(outfile,"Xreal[%d]: %.4f Ximag[%d]: %.4fi\n",i, hipCreal(returnedFFT[i]), i, hipCimag((returnedFFT[i])));
			fprintf(outfile,"================================\n");

		}
	//end iteration loop	
		double timeElapsed = finish-start; //Time for that iteration
		avgtime = avgtime + timeElapsed; //AVG the time 
		fprintf(outfile,"Time Elaspsed on Iteration %d: %f Seconds\n", (h+1),timeElapsed);
		hipFree(tableValues);
		hipFree(returnedFFT);	
	}
	avgtime = avgtime / howmanytimesavg;
	fprintf(outfile,"\nAverage Time Elaspsed: %f Seconds", avgtime);
	fclose(outfile); //close file
	return 0;
}